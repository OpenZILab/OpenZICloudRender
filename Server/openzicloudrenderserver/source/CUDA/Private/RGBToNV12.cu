#include "hip/hip_runtime.h"
//
// Copyright by Cengzi Technology Co., Ltd
// Created by OpenZILab
// DateTime: 2022/07/27 14:18
//

// TODO: 1.texRef->texObj 2.hipBindTextureToArray

#include <stdint.h>

#include "RGBToNV12.cuh"

using uint8 = uint8_t;
using uint32 = uint32_t;
struct FPixel_RGB10A2
{
    uint32 R : 10;
    uint32 G : 10;
    uint32 B : 10;
    uint32 A : 2;
};

__device__ inline float rgb2y(uchar4 c)
{
    return 0.257f * c.x + 0.504f * c.y + 0.098f * c.z + 16.0f;
}

__device__ inline float rgb2u(uchar4 c)
{
    return -0.148f * c.x - 0.291f * c.y + 0.439f * c.z + 128.0f;
}

__device__ inline float rgb2v(uchar4 c)
{
    return 0.439f * c.x - 0.368f * c.y - 0.071f * c.z + 128.0f;
}
__device__ inline float bgr2y(uchar4 c)
{
    // return 0.257f * c.z + 0.504f * c.y + 0.098f * c.x + 16.0f;
    return ((66 * c.z + 129 * c.y + 25 * c.x) >> 8) + 16;
    // return 0.299 * c.z + 0.587 * c.y + 0.114f * c.x ;
}

__device__ inline float bgr2u(uchar4 c)
{
    return -0.148f * c.z - 0.291f * c.y + 0.439f * c.x + 128.0f;
    // return -0.1687 * c.z - 0.3313f * c.y + 0.5f * c.x + 128.0f;
}

__device__ inline float bgr2v(uchar4 c)
{
    return 0.439f * c.z - 0.368f * c.y - 0.071f * c.x + 128.0f;
    // return 0.5 * c.z - 0.4187f * c.y - 0.0813f * c.x + 128.0f;
}
__device__ inline uchar4 rgba10torgba8(uchar4 &c)
{
    FPixel_RGB10A2 *SrcPixelPtr = (FPixel_RGB10A2 *)(&c);
    uchar4 dst;
    dst.x = (uint8)((float)SrcPixelPtr->R * 0.249266f);
    dst.y = (uint8)((float)SrcPixelPtr->G * 0.249266f);
    dst.z = (uint8)((float)SrcPixelPtr->B * 0.249266f);
    dst.w = 255;
    return dst;
}

texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef;
// hipTextureObject_t texObj;

__global__ void RGBA2NV12_kernel(uint8_t *dstImage, size_t destPitch,
                                 uint32_t width, uint32_t height)
{
    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    int32_t x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    int32_t y = blockIdx.y * (blockDim.y << 1) + (threadIdx.y << 1);

    int x1 = x + 1;
    int y1 = y + 1;

    if (x1 >= width)
        return; // x = width - 1;

    if (y1 >= height)
        return; // y = height - 1;

    uchar4 c00 = tex2D(texRef, x, y);
    uchar4 c01 = tex2D(texRef, x1, y);
    uchar4 c10 = tex2D(texRef, x, y1);
    uchar4 c11 = tex2D(texRef, x1, y1);
    c00 = rgba10torgba8(c00);
    c01 = rgba10torgba8(c01);
    c10 = rgba10torgba8(c10);
    c11 = rgba10torgba8(c11);

    uint8_t y00 = (uint8_t)(rgb2y(c00) + 0.5f);
    uint8_t y01 = (uint8_t)(rgb2y(c01) + 0.5f);
    uint8_t y10 = (uint8_t)(rgb2y(c10) + 0.5f);
    uint8_t y11 = (uint8_t)(rgb2y(c11) + 0.5f);

    uint8_t u = (uint8_t)((rgb2u(c00) + rgb2u(c01) + rgb2u(c10) + rgb2u(c11)) * 0.25f + 0.5f);
    uint8_t v = (uint8_t)((rgb2v(c00) + rgb2v(c01) + rgb2v(c10) + rgb2v(c11)) * 0.25f + 0.5f);

    dstImage[destPitch * y + x] = y00;
    dstImage[destPitch * y + x1] = y01;
    dstImage[destPitch * y1 + x] = y10;
    dstImage[destPitch * y1 + x1] = y11;

    uint32_t chromaOffset = destPitch * height;
    int32_t x_chroma = x;
    int32_t y_chroma = y >> 1;

    dstImage[chromaOffset + destPitch * y_chroma + x_chroma] = u;
    dstImage[chromaOffset + destPitch * y_chroma + x_chroma + 1] = v;
}

__global__ void BGRA2NV12_kernel(uint8_t *dstImage, size_t destPitch,
                                 uint32_t width, uint32_t height)
{
    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    int32_t x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    int32_t y = blockIdx.y * (blockDim.y << 1) + (threadIdx.y << 1);

    int x1 = x + 1;
    int y1 = y + 1;

    if (x1 >= width)
        return; // x = width - 1;

    if (y1 >= height)
        return; // y = height - 1;

    uchar4 c00 = tex2D(texRef, x, y);
    uchar4 c01 = tex2D(texRef, x1, y);
    uchar4 c10 = tex2D(texRef, x, y1);
    uchar4 c11 = tex2D(texRef, x1, y1);

    uint8_t y00 = (uint8_t)(bgr2y(c00) + 0.5f);
    uint8_t y01 = (uint8_t)(bgr2y(c01) + 0.5f);
    uint8_t y10 = (uint8_t)(bgr2y(c10) + 0.5f);
    uint8_t y11 = (uint8_t)(bgr2y(c11) + 0.5f);

    uint8_t u = (uint8_t)((bgr2u(c00) + bgr2u(c01) + bgr2u(c10) + bgr2u(c11)) * 0.25f + 0.5f);
    uint8_t v = (uint8_t)((bgr2v(c00) + bgr2v(c01) + bgr2v(c10) + bgr2v(c11)) * 0.25f + 0.5f);

    dstImage[destPitch * y + x] = y00;
    dstImage[destPitch * y + x1] = y01;
    dstImage[destPitch * y1 + x] = y10;
    dstImage[destPitch * y1 + x1] = y11;

    uint32_t chromaOffset = destPitch * height;
    int32_t x_chroma = x;
    int32_t y_chroma = y >> 1;

    dstImage[chromaOffset + destPitch * y_chroma + x_chroma] = u;
    dstImage[chromaOffset + destPitch * y_chroma + x_chroma + 1] = v;
}

__device__ uchar4 GetPixelWithWaterMark(uint32_t x, uint32_t y, uint32_t areaX, uint32_t areaY, uint32_t areaWidth, uint32_t areaHeight, uint8_t *waterMarkImage, size_t waterMarkPitch)
{
    uchar4 Pixel = tex2D(texRef, x, y);

    if (x >= areaX && x <= (areaX + areaWidth) && y >= areaY && y <= (areaY + areaHeight))
    {
        uint32_t locX = x - areaX;
        uint32_t locY = y - areaY;
        uint32_t MarkPixelOffset = (locY * waterMarkPitch * areaWidth) + locX * waterMarkPitch;
        uint8 markR = *(waterMarkImage + MarkPixelOffset);
        uint8 markG = *(waterMarkImage + MarkPixelOffset + 1);
        uint8 markB = *(waterMarkImage + MarkPixelOffset + 2);
        uint8 markA = *(waterMarkImage + MarkPixelOffset + 3);
        Pixel.x = uint8(markR * markA / 255.0 + Pixel.x * (1 - markA / 255.0));
        Pixel.y = uint8(markG * markA / 255.0 + Pixel.y * (1 - markA / 255.0));
        Pixel.z = uint8(markB * markA / 255.0 + Pixel.z * (1 - markA / 255.0));
    }

    return Pixel;
}

__global__ void BGRA2NV12_WithWaterMark_kernel(uint8_t *dstImage, size_t destPitch,
                                               uint32_t width, uint32_t height,
                                               uint8_t *waterMarkImage, size_t waterMarkPitch,
                                               uint32_t markWidth, uint32_t markHeight,
                                               uint32_t markLocationX, uint32_t markLocationY)
{
    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    int32_t x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    int32_t y = blockIdx.y * (blockDim.y << 1) + (threadIdx.y << 1);

    int x1 = x + 1;
    int y1 = y + 1;

    if (x1 >= width)
        return; // x = width - 1;

    if (y1 >= height)
        return; // y = height - 1;

    uchar4 c00 = GetPixelWithWaterMark(x, y, markLocationX, markLocationY, markWidth, markHeight, waterMarkImage, waterMarkPitch);
    uchar4 c01 = GetPixelWithWaterMark(x1, y, markLocationX, markLocationY, markWidth, markHeight, waterMarkImage, waterMarkPitch);
    uchar4 c10 = GetPixelWithWaterMark(x, y1, markLocationX, markLocationY, markWidth, markHeight, waterMarkImage, waterMarkPitch);
    uchar4 c11 = GetPixelWithWaterMark(x1, y1, markLocationX, markLocationY, markWidth, markHeight, waterMarkImage, waterMarkPitch);

    uint8_t y00 = (uint8_t)(bgr2y(c00) + 0.5f);
    uint8_t y01 = (uint8_t)(bgr2y(c01) + 0.5f);
    uint8_t y10 = (uint8_t)(bgr2y(c10) + 0.5f);
    uint8_t y11 = (uint8_t)(bgr2y(c11) + 0.5f);

    uint8_t u = (uint8_t)((bgr2u(c00) + bgr2u(c01) + bgr2u(c10) + bgr2u(c11)) * 0.25f + 0.5f);
    uint8_t v = (uint8_t)((bgr2v(c00) + bgr2v(c01) + bgr2v(c10) + bgr2v(c11)) * 0.25f + 0.5f);

    dstImage[destPitch * y + x] = y00;
    dstImage[destPitch * y + x1] = y01;
    dstImage[destPitch * y1 + x] = y10;
    dstImage[destPitch * y1 + x1] = y11;

    uint32_t chromaOffset = destPitch * height;
    int32_t x_chroma = x;
    int32_t y_chroma = y >> 1;

    dstImage[chromaOffset + destPitch * y_chroma + x_chroma] = u;
    dstImage[chromaOffset + destPitch * y_chroma + x_chroma + 1] = v;
}

extern "C" hipError_t RGBA2NV12(hipArray *srcImage,
                                 uint8_t *dstImage, size_t destPitch,
                                 uint32_t width, uint32_t height)
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);

    // Set texture parameters
    texRef.addressMode[0] = hipAddressModeWrap;
    texRef.addressMode[1] = hipAddressModeWrap;
    texRef.filterMode = hipFilterModePoint;
    texRef.normalized = false;

    hipError_t cudaStatus = hipBindTextureToArray(texRef, srcImage, channelDesc);
    if (cudaStatus != hipSuccess)
    {
        return cudaStatus;
    }

    dim3 block(32, 16, 1);
    dim3 grid((width + (2 * block.x - 1)) / (2 * block.x), (height + (2 * block.y - 1)) / (2 * block.y), 1);

    RGBA2NV12_kernel<<<grid, block>>>(dstImage, destPitch, width, height);

    hipDeviceSynchronize();

    cudaStatus = hipGetLastError();
    return cudaStatus;
}

extern "C" hipError_t BGRA2NV12(hipArray *srcImage,
                                 uint8_t *dstImage, size_t destPitch,
                                 uint32_t width, uint32_t height)
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);

    // Set texture parameters
    texRef.addressMode[0] = hipAddressModeWrap;
    texRef.addressMode[1] = hipAddressModeWrap;
    texRef.filterMode = hipFilterModePoint;
    texRef.normalized = false;

    hipError_t cudaStatus = hipBindTextureToArray(texRef, srcImage, channelDesc);
    if (cudaStatus != hipSuccess)
    {
        return cudaStatus;
    }

    dim3 block(32, 16, 1);
    dim3 grid((width + (2 * block.x - 1)) / (2 * block.x), (height + (2 * block.y - 1)) / (2 * block.y), 1);

    BGRA2NV12_kernel<<<grid, block>>>(dstImage, destPitch, width, height);

    hipDeviceSynchronize();

    cudaStatus = hipGetLastError();
    return cudaStatus;
}

extern "C" hipError_t BGRA2NV12_WithWaterMark(hipArray *srcImage,
                                               uint8_t *dstImage, size_t destPitch,
                                               uint32_t width, uint32_t height,
                                               uint8_t *waterMarkImage, size_t waterMarkPitch,
                                               uint32_t markWidth, uint32_t markHeight,
                                               uint32_t markLocationX, uint32_t markLocationY)
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);

    // Set texture parameters
    texRef.addressMode[0] = hipAddressModeWrap;
    texRef.addressMode[1] = hipAddressModeWrap;
    texRef.filterMode = hipFilterModePoint;
    texRef.normalized = false;
    uint8_t* WaterMarkPtr;
    hipMalloc((void **)&WaterMarkPtr, markWidth*markHeight*waterMarkPitch);
    hipMemcpy(WaterMarkPtr, waterMarkImage, markWidth*markHeight*waterMarkPitch, hipMemcpyHostToDevice);
    hipError_t cudaStatus = hipBindTextureToArray(texRef, srcImage, channelDesc);
    if (cudaStatus != hipSuccess)
    {
        hipFree(WaterMarkPtr);
        return cudaStatus;
    }
    dim3 block(32, 16, 1);
    dim3 grid((width + (2 * block.x - 1)) / (2 * block.x), (height + (2 * block.y - 1)) / (2 * block.y), 1);

    BGRA2NV12_WithWaterMark_kernel<<<grid, block>>>(dstImage, destPitch, width, height, WaterMarkPtr, waterMarkPitch, markWidth, markHeight, markLocationX, markLocationY);

    hipDeviceSynchronize();
    hipFree(WaterMarkPtr);

    cudaStatus = hipGetLastError();
    return cudaStatus;
}